
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <fstream>

// #define RANDOM_WALK

typedef struct {
    float x;
    float y;
    float horizontal_speed;
    float vertical_speed;
    uint seed;
} Particle;

typedef struct {
    int x;
    int y;
} Veci2D;

const float radius = 2.0f;
const int ceil_radius = (int)radius + ((((float)(int)radius) < radius) ? 1 : 0);
const float max_speed = 3.0f;
const int particle_count = 4096 * 8;

const int grid_size = 1024 * 8;
const int grid_width = grid_size;
const int grid_height = grid_size;

__device__ int grid[grid_height][grid_width];
__constant__ Veci2D* circle_indices;
__constant__ int circle_indices_length;
__device__ int border_left;
__device__ int border_right;
__device__ int border_top;
__device__ int border_bottom;

void VecAdd();
void simulate();
void tick(Particle* particles, int tick_count);
__device__ int random_int(int min, int max, uint seed);

#define print(message) std::cout << message << std::endl

int main() {
    print("starting");
    simulate();
    print("done");
}

// checks for cuda errors
// could be improved
void cuda_error() {
    auto result = hipGetLastError();
    if (result != hipSuccess) {
        std::cout << "error: " << result << std::endl;
    }
    else {
        std::cout << "success" << std::endl;
    }
}

// sets the grid values to -1
__global__ void init_grid_negative() {
    grid[blockIdx.y * blockDim.y + threadIdx.y][blockIdx.x * blockDim.x + threadIdx.x] = -1;
}

// sets the center of the grid to 0
__global__ void init_grid_center() {
    grid[grid_height / 2][grid_width / 2] = 0;
    border_top = grid_height / 2;
    border_bottom = grid_height / 2;
    border_left = grid_width / 2;
    border_right = grid_width / 2;
}

// outputs the grid (and its widht/height) to a file
void output_grid() {
    // get grid from GPU memory
    size_t mem_size = sizeof(int) * grid_height * grid_width;
    int* host_grid = (int*)malloc(mem_size);
    hipMemcpyFromSymbol(host_grid, HIP_SYMBOL(grid), mem_size, 0, hipMemcpyDeviceToHost);

    // create file
    std::ofstream output_file;
    output_file.open("grid_output.bin", std::ios::binary);
    if(output_file.is_open()) {
        print("output_file is open");
    }

    // output to file
    const int ints[2] = {grid_width, grid_height};
    output_file.write((const char*) &ints, sizeof(int) * 2);
    //output_file.write((const char*) &grid_height, sizeof(int));
    output_file.write((const char*) host_grid, mem_size);


    // std::cout << std::endl << std::endl << "[";
    // for(int y = 0; y < grid_height; y++) {
    //     std::cout << "\"";
    //     for(int x = 0; x < grid_width; x++) {
    //         int value_at_xy = *(host_grid + x + y * grid_width);
    //         // std::cout << value_at_xy << ",";
    //         //std::cout << (value_at_xy >= 0) ? "1" : "0";
    //         print(value_at_xy);
    //     } 
    //     std::cout << "\"," << std::endl;
    // }
    // std::cout << "]" << std::endl << std::endl;
    
    // clean up
    output_file.close();
    delete host_grid;
}

__device__ uint hash(uint x) {
    const uint seed = 1324567967 + 2;
    x += seed;
    x = ((x >> 16) ^ x) * seed;
    x = ((x >> 16) ^ x) * seed;
    x = (x >> 16) ^ x;
    return x;
}

// returns an int in the range [min, max) based on seed
__device__ int random_int(int min, int max, uint seed) {
    uint random = hash(seed);
    random %= max - min;
    
    return (int)random + min;
}

// returns a float in the range [0, 1) based on seed;
__device__ float random_float(uint seed) {
    const int max = 10000000;
    int base = random_int(0, max, seed);

    return (float)base / (float)max;
}

__device__ void randomize_speed(Particle* particle, int direction_seed, int speed_seed) {
    float direction = M_PI * 2.0f * random_float(direction_seed);
    float speed = random_float(speed_seed) * max_speed;

    particle->vertical_speed = cosf(direction) * speed;
    particle->horizontal_speed = sinf(direction) * speed;
}

// randomizes all fields of the particle
__device__ void randomize_particle(Particle* particle) {
    uint seed = particle->seed;
    int center_width = border_right - border_left;
    int center_height = border_bottom - border_top;

    particle->x = random_int(0, grid_width - center_width, seed + 0);
    particle->y = random_int(0, grid_height - center_height, seed + 1);
    if(particle->x > border_left) {
        particle->x += center_width;
    }
    if(particle->y > border_top) {
        particle->y += center_height;
    }

    randomize_speed(particle, seed + 2, seed + 3);

    particle->seed = hash(seed);
}

// initializes the particle
__global__ void init_particles(Particle* particles) {
    int i = threadIdx.x + blockIdx.x * blockDim.x; // particle index in the particles array
    Particle* particle = particles + i;
    particle->seed = (uint)i * 4;
    randomize_particle(particle);
}

// prints border_left, border_right, border_top and border_bottom to stdio
void print_boundaries() {
    int left, right, top, bottom;

    hipMemcpyFromSymbol(&left, HIP_SYMBOL(border_left), sizeof(int));
    hipMemcpyFromSymbol(&right, HIP_SYMBOL(border_right), sizeof(int));
    hipMemcpyFromSymbol(&top, HIP_SYMBOL(border_top), sizeof(int));
    hipMemcpyFromSymbol(&bottom, HIP_SYMBOL(border_bottom), sizeof(int));
    print(left << ", " << right << ", " << top << ", " << bottom);
}

void simulate() {
    // initialize grid
    dim3 threadsPerBlock(16, 16);
    dim3 blocks(grid_width / threadsPerBlock.x, grid_height / threadsPerBlock.y);
    init_grid_negative<<<blocks, threadsPerBlock>>>();
    init_grid_center<<<1, 1>>>();

    // initialize particles
    size_t mem_size = particle_count * sizeof(Particle);
    Particle* particles;
    hipMalloc(&particles, mem_size);
    const int particle_threads_per_block = 256;
    const int particle_blocks = particle_count / particle_threads_per_block;

    cuda_error();
    init_particles<<<particle_blocks, particle_threads_per_block>>>(particles);
    // done intializing particles

    print_boundaries();
    cuda_error();

    int tick_count = 0;
    for(int i = 0; true; i++) {
        tick(particles, ++tick_count);

      
        int left, right, top, bottom;

        hipMemcpyFromSymbol(&left, HIP_SYMBOL(border_left), sizeof(int));
        hipMemcpyFromSymbol(&right, HIP_SYMBOL(border_right), sizeof(int));
        hipMemcpyFromSymbol(&top, HIP_SYMBOL(border_top), sizeof(int));
        hipMemcpyFromSymbol(&bottom, HIP_SYMBOL(border_bottom), sizeof(int));

        if(i % 100 == 0) {
            print(left << ", " << right << ", " << top << ", " << bottom);
        }
        const int margin = 50;
        if(left < margin || right > grid_width - margin || top < margin || bottom > grid_height - margin) {
            break;
        }
    }
    cuda_error();
    output_grid();

    hipFree(particles);
}

__global__ void particle_step(Particle* particles, int tick_count) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // particle index in the particles array
    Particle* particle = particles + i;
    
    #ifdef RANDOM_WALK
    // randomize direction and speed
    randomize_speed(particle, particle->seed, particle->seed + 1);
    particle->seed = hash(particle->seed);
    #endif

    // move particle
    particle->x += particle->horizontal_speed;
    particle->y += particle->vertical_speed;

    // check bounds
    if(particle->x - radius <= 0.0f) {
        particle->x = 0.01f + radius;
        particle->horizontal_speed *= -1.0f;
    }
    else if(particle->x + radius >= grid_width) {
        particle->x = grid_width - 0.01f - radius;
        particle->horizontal_speed *= -1.0f;
    }
    if(particle->y - radius <= 0.0f) {
        particle->y = 0.01f + radius ;
        particle->vertical_speed *= -1.0f;
    }
    else if(particle->y + radius >= grid_height) {
        particle->y = grid_height - 0.01f - radius;
        particle->vertical_speed *= -1.0f;
    }

    // calculate some variable values to be used later
    const int diameter = ceil_radius * 2;

    int left = (int)(particle->x - radius);
    int top = (int)(particle->y - radius);
    float modulo_x = fmod(particle->x, 1.0f);
    float modulo_y = fmod(particle->y, 1.0f);
    bool looping = true;

    for(int dx = -ceil_radius; dx <= ceil_radius && looping; dx++) {
        for(int dy = -ceil_radius; dy <= ceil_radius && looping; dy++) {
            // calculate distance from center of the particle
            float distance_x = -dx + modulo_x;
            float distance_y = -dy + modulo_y;

            if(distance_x * distance_x + distance_y * distance_y < radius * radius) {
                // position is within radius of the center
                if(grid[(int)(particle->y - distance_y)][(int)(particle->x - distance_x)] >= 0) {
                    // hit another particle

                    for(int dx2 = -ceil_radius; dx2 <= ceil_radius; dx2++) {
                        for(int dy2 = -ceil_radius; dy2 <= ceil_radius; dy2++) {
                            // calculate distance from center of the particle
                            float distance_x2 = -dx2 + modulo_x;
                            float distance_y2 = -dy2 + modulo_y;
                
                            if(distance_x2 * distance_x2 + distance_y2 * distance_y2 < radius * radius) {
                                // calculate position in grid
                                int absolute_x = (int)(particle->x - distance_x2);
                                int absolute_y = (int)(particle->y - distance_y2);
                    
                                // if the absolute_x/y are within the grid
                                if(absolute_x >= 0 && absolute_x < grid_width && absolute_y >= 0 && absolute_y < grid_height) {
                                    // set the grid to being hit
                                    grid[absolute_y][absolute_x] = tick_count;

                                    /*
                                        Because the program writes and reads from the same grid in a single tick,
                                        the algorithm isn't completely deterministic. I could use two different 
                                        grids and then copy values, but it doesn't feel necessary.
                                    */
                                }
                            }
                        }
                    }
                    
                    atomicMin(&border_left, (int)(particle->x - radius));
                    atomicMax(&border_right, (int)(particle->x + radius));
                    atomicMin(&border_top, (int)(particle->y - radius));
                    atomicMax(&border_bottom, (int)(particle->y + radius));
                    
                    // give the particle a random new position and speed
                    randomize_particle(particle);

                    looping = false;
                    break;
                }
            }
            
        }
    }
}

// perform one tick
void tick(Particle* particles, int tick_count) {
    const int threads_per_block = 32;
    const int blocks = particle_count / threads_per_block;

    particle_step<<<blocks, threads_per_block>>>(particles, tick_count);
}